#include <iostream>
#include <cstdio>
#include <cstdlib>
#include <vector>
#include <ctime>
#include "hip/hip_runtime.h"
#include <cfloat>

#define BLOCKSIZE 1024

/**
  * Version 1: Ecrire un kernel GPU 1D qui trouve l'element minimum d'un tableau dA[N] pour chaque bloc et ecrit le minimum de chaque bloc dans une case de dAmin. En suite, CPU reprend dAmin et calcul le minimum global en sequentiel sur ce petit tableau.
  *
  * Version 2: Le premier appel au findMinimum reduit la taille du tableau a parcourir en sequentiel a N/BLOCKSIZE. Dans cette version, utiliser findMinimum deux fois a la suite afin de reduire la taille du tableau a parcourir en sequentiel a N/(BLOCKSIZE*BLOCKSIZE) (pour que la partie sequentielle en CPU devient vraiment negligable).
  *
  * Pour trouver le minimum des deux flottants en GPU, utiliser la fonction fminf(x, y)
  */

__global__ void findMinimum(float *dA, float *dAmin, int N)
{
  __shared__ volatile float buff[BLOCKSIZE];
  int idx = threadIdx.x + blockIdx.x * BLOCKSIZE;
  // A FAIRE ...
}

using namespace std;

int main()
{
  srand(1234);
  int N = 100000000;
  int numBlocks;// = ???; (A FAIRE ...)
  float *A, *dA; // Le tableau dont minimum on va chercher
  float *Amin, *dAmin; // Amin contiendra en suite le tableau reduit par un facteur de BLOCKSIZE apres l'execution du kernel GPU

  // Allour les tableaux A[N] et Amin[numBlocks] de maniere ``pined'' sur le CPU
  // Allouer les tableaux dA[N] et dAmin[numBlocks] sur le GPU
  // A FAIRE ...

  // Initialiser le tableau A
  for (int i = 0; i < N; i++) { A[i] = (float)(rand() % 1000); }
  A[rand() % N] = -1.0; // Mettre le minimum a -1.

  // Mettre A sur le GPU (dA) avec memcpy
  // A FAIRE ...

  float minA = FLT_MAX; // Affecter le maximum float a minA
  // Trouver le minimum du tableau dA, mettre dAmin dans le CPU, puis trouver le minimum global et le mettre dans la variable minA
  // A FAIRE ...
  // findMinimum<<<...>>>(...)
  // ...

  // Verifier le resultat
  if (minA == -1) { cout << "The minimum is correct!" << endl; }
  else { cout << "The minimum found (" << minA << ") is incorrect (it should have been -1)!" << endl; }

  return 0;
}
